/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) {
  hipDeviceProp_t dP;
  float min_cc = 6.0;

  int rc = hipGetDeviceProperties(&dP, 0);
  if (rc != hipSuccess) {
    hipError_t error = hipGetLastError();
    printf("CUDA error: %s", hipGetErrorString(error));
    return rc; /* Failure */
  }
  if ((dP.major + (dP.minor / 10)) < min_cc) {
    printf("Min Compute Capability of %2.1f required:  %d.%d found", min_cc, dP.major, dP.minor);
    return 1; /* Failure */
  } else {
    printf("%d%d", dP.major, dP.minor);
    return 0; /* Success */
  }
}
